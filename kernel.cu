
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512

__device__ unsigned int warpDistribution[33] = {0};

__device__ void countWarpDistribution(){

      unsigned int mask = __popc(__activemask());
      atomicAdd(&warpDistribution[mask],1);

}

__device__ void printWarpDistribution(){
    printf("\n Warp Distribution: \n");
    for(int i = 0; i < 33; i++){
        printf("W%d: %u, ",i,warpDistribution[i]);
    }
    printf("\n\n");
}

__global__ void naiveReduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    // NAIVE REDUCTION IMPLEMENTATION

      
    if(threadIdx.x == 0 && blockIdx.x == 0)
      printWarpDistribution();  
}

__global__ void optimizedReduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    // OPTIMIZED REDUCTION IMPLEMENTATION

    if(threadIdx.x == 0 && blockIdx.x == 0)
      printWarpDistribution();  
}
